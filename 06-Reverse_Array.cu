#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float temp;
    if (idx < N / 2) {
        temp = input[idx];
        input[idx] = input[N-idx-1];
        input[N-idx-1] = temp;
    }
}

// input is device pointer
void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}
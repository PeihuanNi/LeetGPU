#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_y = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx_x < cols && idx_y < rows) {
        output[idx_x * rows + idx_y] = input[idx_y * cols + idx_x];
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}